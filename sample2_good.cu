
#include <hip/hip_runtime.h>
/* Correct Code For Tower Of Hanoi */

int transfer(int n, int from, int to, int through);

int main(){
	transfer(6, 'A', 'C', 'B');

	return 0;
}

int transfer(int n, int from, int to, int through){
	if(n==1){
		printf("%d: %c -> %c\n", n, from, to);
		return;
	}
	
	transfer(n=n-1, from, through, to);

	printf("%d: %c -> %c\n", n+1, from, to);

	transfer(n, through, to, from);
}